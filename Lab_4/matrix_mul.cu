#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "matrix_mul.cuh"

__global__ void matrixMulCUDA(const float* A, const float* B, float* C, int N) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < N && col < N) {
        float sum = 0.0f;
        for (int k = 0; k < N; ++k) {
            sum += A[row * N + k] * B[k * N + col];
        }
        C[row * N + col] = sum;
    }
}

void runCudaMul(const float* h_A, const float* h_B, float* h_C, int N,
    int blockSizeX, int blockSizeY) {
    size_t size = N * N * sizeof(float);
    float* d_A, * d_B, * d_C;

    hipMalloc((void**)&d_A, size);
    hipMalloc((void**)&d_B, size);
    hipMalloc((void**)&d_C, size);

    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    dim3 block(blockSizeX, blockSizeY);
    dim3 grid((N + block.x - 1) / block.x, (N + block.y - 1) / block.y);
    matrixMulCUDA << <grid, block >> > (d_A, d_B, d_C, N);
    hipDeviceSynchronize();

    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}

